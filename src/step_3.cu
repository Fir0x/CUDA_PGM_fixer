#include "hip/hip_runtime.h"
#include "fix_gpu.hh"

#include <iostream>
#include <thrust/binary_search.h>
#include <thrust/find.h>

namespace Core
{

    struct non_zero_functor
    {
        __device__ __host__ bool operator()(const int x)
        {
            return x != 0;
        }
    };

    struct apply_functor
    {
        thrust::device_ptr<int> histogram;
        const int cdf_min;
        const size_t image_size;

        apply_functor(thrust::device_ptr<int> _histogram, int _cdf_min, size_t _image_size)
            : histogram(_histogram), cdf_min(_cdf_min), image_size(_image_size)
        {
        }

        __device__ __host__ float operator()(const int pixel)
        {
            return roundf(((histogram[pixel] - cdf_min) / static_cast<float>(image_size - cdf_min)) * 255.0f);
        }
    };

    // Histogram equalization
    void step_3(thrust::device_vector<int> &to_fix, size_t image_size)
    {
        // 1. Histogram
        thrust::device_vector<int> histogram(256, 0);
        thrust::device_vector<int> to_fix_tmp(to_fix.begin(), to_fix.begin() + image_size);
        thrust::sort(to_fix_tmp.begin(), to_fix_tmp.end());

        thrust::counting_iterator<int> search_begin(0);
        thrust::upper_bound(to_fix_tmp.begin(), to_fix_tmp.end(),
                            search_begin, search_begin + 256,
                            histogram.begin());

        thrust::adjacent_difference(histogram.begin(), histogram.end(),
                                    histogram.begin());

        // 2. Compute the inclusive sum scan of the histogram
        thrust::inclusive_scan(thrust::device, histogram.begin(), histogram.end(), histogram.begin());

        // 3. Find the first non-zero value in the cumulative histogram
        auto first_none_zero = thrust::find_if(thrust::device, histogram.begin(), histogram.end(), non_zero_functor());
        const int cdf_min = *first_none_zero;

        // 4. Apply the map transformation of the histogram equalization
        apply_functor apply_instance(histogram.data(), cdf_min, image_size);

        thrust::transform(to_fix.begin(), to_fix.begin() + image_size, to_fix.begin(), apply_instance);
    }
} // namespace Core

namespace CustomCore
{
    __global__ void build_histogram(int *to_fix, int *histo, int size)
    {
        int id = blockIdx.x * blockDim.x + threadIdx.x;
        if (id < size)
        {
            int index_histo = to_fix[id];
            // TODO Easy to improve
            atomicAdd(histo + index_histo, 1);
        }
    }

    __global__ void find_first_non_zero(int *histo, int work_per_thread, int *find_first_non_zero)
    {
        __shared__ int min_non_zeros[16];
        
        int start_id = threadIdx.x * work_per_thread;
        int min = histo[start_id]; 

        // Will iterate on a part of the array
        // Separe te work on x thread group
        for (int i = start_id + 1; i < start_id + work_per_thread; i++)
        {
            int val = histo[i];
            if (min == 0 || (val != 0 && val < min))
            {
                min = val;
            }
        }
        min_non_zeros[threadIdx.x] = min;

        __syncthreads();
        
        min = 0;
        // A thread alone is here to get the final result
        if (threadIdx.x == 0)
        {
            for (int i = 0; i < NB_THREADS / work_per_thread; i++)
            {
                int val = min_non_zeros[i];
                if (min == 0 || (val != 0 && val < min))
                {
                    min = val;
                }
            }
            *find_first_non_zero = min;
        }
    }

    __global__ void histo_equalization(int *to_fix, int *histo, int *first_non_zero, int size)
    {
        int id = blockIdx.x * blockDim.x + threadIdx.x;
        if (id < size)
        {
            int fix_val = to_fix[id];
            int histo_val = histo[fix_val];
            to_fix[id] = std::roundf(((histo_val - *first_non_zero) / static_cast<float>(size - *first_non_zero)) * 255.0f);
        }
    }
    void step_3([[maybe_unused]] int *to_fix, [[maybe_unused]] ImageInfo imageInfo)
    {
        //std::cout << "=== Start step 3 custom" << std::endl;
        int size = imageInfo.height * imageInfo.width;
        int nbBlocks = std::ceil((float)size / NB_THREADS);

        // 1. Histogram
        int *histogram;
        cudaMalloc_custom(&histogram, sizeof(int) * 256);
        hipMemset(histogram, 0, sizeof(int) * 256);
        build_histogram<<<nbBlocks, NB_THREADS>>>(to_fix, histogram, size);
        checkKernelError("build_histogram");
        //hipDeviceSynchronize();

        // { // debug
        //     thrust::device_ptr<int> tmp_histogram = thrust::device_pointer_cast(histogram);
        //     std::cout << "Histogram accumulation: " << thrust::reduce(tmp_histogram, tmp_histogram + 256, 0) << std::endl;
        // }

        // 2. Compute the inclusive sum scan of the histogram
        scan(histogram, 256, true);

        // 3. Find the first non-zero value in the cumulative histogram
        int *first_non_zero;
        cudaMalloc_custom(&first_non_zero, sizeof(int));
        int work_per_thread = 16;
        find_first_non_zero<<<1, 16>>>(histogram, work_per_thread, first_non_zero);
        checkKernelError("find_first_non_zero");
        //hipDeviceSynchronize();

        // { // debug
        //     thrust::device_ptr<int> tmp_first = thrust::device_pointer_cast(first_non_zero);
        //     std::cout << "First cdf_min: " << *tmp_first << std::endl;
        // }

        // 4. Apply the map transformation of the histogram equalization
        histo_equalization<<<nbBlocks, NB_THREADS>>>(to_fix, histogram, first_non_zero, size);
        checkKernelError("histo_equalization");
        //hipDeviceSynchronize();

        // { // debug
        //     thrust::device_ptr<int> tmp_fix = thrust::device_pointer_cast(to_fix);
        //     std::cout << "Last accumulation: " << thrust::reduce(tmp_fix, tmp_fix + size, 0) << std::endl;
        // }

        hipFree(histogram);
        hipFree(first_non_zero);
    }
} // namespace CustomCore