#include "hip/hip_runtime.h"
#include "fix_gpu.hh"

#include <iostream>
#include <thrust/binary_search.h>
#include <thrust/find.h>

namespace Core
{

    struct non_zero_functor
    {
        __device__ __host__ bool operator()(const int x)
        {
            return x != 0;
        }
    };

    struct apply_functor
    {
        thrust::device_ptr<int> histogram;
        const int cdf_min;
        const size_t image_size;

        apply_functor(thrust::device_ptr<int> _histogram, int _cdf_min, size_t _image_size)
            : histogram(_histogram), cdf_min(_cdf_min), image_size(_image_size)
        {
        }

        __device__ __host__ float operator()(const int pixel)
        {
            return roundf(((histogram[pixel] - cdf_min) / static_cast<float>(image_size - cdf_min)) * 255.0f);
        }
    };

    // Histogram equalization
    void step_3(thrust::device_vector<int> &to_fix, size_t image_size)
    {
        // 1. Histogram
        thrust::device_vector<int> histogram(256, 0);
        thrust::device_vector<int> to_fix_tmp(to_fix.begin(), to_fix.begin() + image_size);
        thrust::sort(to_fix_tmp.begin(), to_fix_tmp.end());

        thrust::counting_iterator<int> search_begin(0);
        thrust::upper_bound(to_fix_tmp.begin(), to_fix_tmp.end(),
                            search_begin, search_begin + 256,
                            histogram.begin());

        thrust::adjacent_difference(histogram.begin(), histogram.end(),
                                    histogram.begin());

        // 2. Compute the inclusive sum scan of the histogram
        thrust::inclusive_scan(thrust::device, histogram.begin(), histogram.end(), histogram.begin());

        // 3. Find the first non-zero value in the cumulative histogram
        auto first_none_zero = thrust::find_if(thrust::device, histogram.begin(), histogram.end(), non_zero_functor());
        const int cdf_min = *first_none_zero;

        // 4. Apply the map transformation of the histogram equalization
        apply_functor apply_instance(histogram.data(), cdf_min, image_size);

        thrust::transform(to_fix.begin(), to_fix.begin() + image_size, to_fix.begin(), apply_instance);
    }
} // namespace Core

namespace CustomCore
{
    __global__ void build_histogram(int *to_fix, int *histo, int size)
    {
        int id = blockIdx.x * blockDim.x + threadIdx.x;
        if (id < size)
        {
            int index_histo = to_fix[id];
            if (index_histo < 0 || index_histo >= 256) 
                index_histo = 0;
                //std::cout << "Error index " << index_histo << std::endl;
            else 
                histo[index_histo] += 1;
        }
    }

    __global__ void find_first_non_zero(int *histo, int work_per_thread, int *find_first_non_zero)
    {
        __shared__ int min_non_zeros[16];
        int start_id = threadIdx.x * work_per_thread;

        // Will iterate on a part of the array
        // Separe te work on x thread group
        for (int i = start_id; i < start_id + work_per_thread; i++)
        {
            if (histo[i] != 0)
            {
                min_non_zeros[threadIdx.x] = i;
                break;
            }
        }

        __syncthreads();

        // A thread alone is here to get the final result
        if (threadIdx.x == 0)
        {
            for (int i = 0; i < NB_THREADS / work_per_thread; i++)
            {
                int real_min = min_non_zeros[i];
                if (real_min != -1)
                {
                    *find_first_non_zero = real_min;
                    break;
                }
            }
        }
    }

    __global__ void histo_equalization(int* to_fix, int* histo, int* first_non_zero, int size) 
    {
        int id = blockIdx.x * blockDim.x + threadIdx.x;
        if (id < size)
        {
            int fix_val = to_fix[id];
            int histo_val = histo[fix_val];
            to_fix[id] = std::roundf(((histo_val - *first_non_zero) / static_cast<float>(size - *first_non_zero)) * 255.0f);
        }
    }
    void step_3([[maybe_unused]] int *to_fix, [[maybe_unused]] ImageInfo imageInfo)
    {
        std::cout << "Step 3 custom" << std::endl;
        int size = imageInfo.height * imageInfo.width;
        int nbBlocks = std::ceil((float)size / NB_THREADS);

        // 1. Histogram
        int *histogram;
        cudaMalloc_custom(&histogram, sizeof(int) * 256);
        hipMemset(histogram, 0, sizeof(int) * 256);
        build_histogram<<<nbBlocks, NB_THREADS>>>(to_fix, histogram, size);
        checkKernelError("build_histogram");
        hipDeviceSynchronize();
        std::cout << "End histogram" << std::endl;
        
        // 2. Compute the inclusive sum scan of the histogram
        scan(to_fix, size, true);
        std::cout << "End scan" << std::endl;

        // 3. Find the first non-zero value in the cumulative histogram
        int *first_non_zero;
        cudaMalloc_custom(&first_non_zero, sizeof(int));
        int work_per_thread = 16;
        find_first_non_zero<<<1, 16>>>(histogram, work_per_thread, first_non_zero);
        checkKernelError("find_first_non_zero");
        hipDeviceSynchronize();
        //std::cout << "First cdf_min: " << *first_non_zero << std::endl;

        // 4. Apply the map transformation of the histogram equalization
        histo_equalization<<<nbBlocks, NB_THREADS>>>(to_fix, histogram, first_non_zero, size);
        checkKernelError("histo_equalization");
        std::cout << "End histo equalization" << std::endl;
        hipDeviceSynchronize();

        hipFree(histogram);
        //hipFree(first_non_zero);
    }
} // namespace CustomCore