#include "fix_gpu.hh"
#include <thrust/scan.h>
#include <thrust/execution_policy.h>

void fix_image_gpu(Image& to_fix)
{
    // Send image to GPU 
    thrust::device_vector<int> d_fix (to_fix.buffer);
    
    Core::step_1(d_fix);
    Core::step_2(d_fix);
    Core::step_3(d_fix);

    // Get data back to CPU
    thrust::copy(to_fix.buffer.begin(), to_fix.buffer.end(), d_fix.begin());
}

void fix_image_gpu_custom(Image& to_fix)
{
    // Send image to GPU 
    const int image_size = to_fix.width * to_fix.height;
    int* image_data; 
    size_t length_pitch; 

    //hipError_t err = hipMallocPitch(&image_data, &length_pitch, sizeof(int) * to_fix.width, to_fix.height);
    hipError_t err = hipMalloc(&image_data, sizeof(int) * to_fix.width * to_fix.height);
    if (err != 0)
        exit(err);
    //hipMemcpy2D(image_data, length_pitch, to_fix.buffer.data(), 0, to_fix.width * sizeof(int), to_fix.height, hipMemcpyHostToDevice);
    hipMemcpy(image_data, to_fix.buffer.data(), to_fix.buffer.size(), hipMemcpyHostToDevice);

    CustomCore::ImageInfo imageInfo = { to_fix.width, to_fix.height, length_pitch };

    CustomCore::step_1(image_data, imageInfo);
    CustomCore::step_2(image_data, imageInfo);
    CustomCore::step_3(image_data, imageInfo);

    // Get data back to CPU
    //hipMemcpy2D(to_fix.buffer.data(), 0, image_data, length_pitch, to_fix.width * sizeof(int), to_fix.height, hipMemcpyDeviceToHost);
    hipMemcpy(to_fix.buffer.data(), image_data, to_fix.buffer.size(), hipMemcpyDeviceToHost);
}