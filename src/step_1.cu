#include "hip/hip_runtime.h"
#include "fix_gpu.hh"

#include <iostream>
#include <thrust/scan.h>
#include <thrust/execution_policy.h>
#include <thrust/copy.h>

namespace Core
{
    struct generate_mask
    {
        __host__ __device__ bool operator()(const int x)
        {
            return x != -27;
        }
    };

    // Compact
    void step_1(thrust::device_vector<int> &to_fix)
    {
        thrust::copy_if(to_fix.begin(), to_fix.end(), to_fix.begin(), generate_mask());
    }

} // namespace Core

namespace CustomCore
{
    __global__ void build_predicate(int *to_fix, int *predicate, int size)
    {
        int id = blockIdx.x * blockDim.x + threadIdx.x;
        if (id < size)
            predicate[id] = to_fix[id] != -27;
    }

    __global__ void scatter(int *to_fix, int *to_fix_cpy, int *predicate, int size)
    {
        int id = blockIdx.x * blockDim.x + threadIdx.x;
        if (id < size)
        {
            int new_val = to_fix_cpy[id];
            if (new_val != -27)
            {
                int new_index = predicate[id];
                to_fix[new_index] = new_val;
            }
        }
    }

    // Compact
    void step_1([[maybe_unused]] int *to_fix, [[maybe_unused]] ImageInfo imageInfo)
    {
        std::cout << "Step 1 custom" << std::endl;

        int size = imageInfo.corrupted_size;
        int nbBlocks = std::ceil((float)size / NB_THREADS);

        // TODO /!\ for first version, don't use streams, will add it later

        // 1 Build the predicate vector
        int *predicate;
        cudaMalloc_custom(&predicate, sizeof(int) * size);
        std::cout << "Start predicate kernel" << std::endl;
        build_predicate<<<nbBlocks, NB_THREADS>>>(to_fix, predicate, size);
        checkKernelError("build_predicate");
        hipDeviceSynchronize();

        // 2 Exclusive sum of the predicate
        std::cout << "Start scan" << std::endl;
        scan(predicate, size, true);

        // 3 Scatter to the corresponding addresses
        const int image_size = imageInfo.width * imageInfo.height;
        { // debug
            thrust::device_ptr<int> tmp_fix = thrust::device_pointer_cast(to_fix);
            auto it = thrust::find(tmp_fix, tmp_fix + size, -27);
            std::cout << "It info before: S " << it - tmp_fix << " E " << tmp_fix + size - it << std::endl;
        }

        int *to_fix_cpy;
        cudaMalloc_custom(&to_fix_cpy, sizeof(int) * size);
        hipMemcpy(to_fix_cpy, to_fix, sizeof(int) * size, hipMemcpyDeviceToDevice);
        std::cout << "Start scatter" << std::endl;
        scatter<<<nbBlocks, NB_THREADS>>>(to_fix, to_fix_cpy, predicate, size);
        checkKernelError("scatter");
        hipDeviceSynchronize();

        { // debug
            thrust::device_ptr<int> tmp_fix = thrust::device_pointer_cast(to_fix);
            auto it = std::find(tmp_fix, tmp_fix + size, -27);
            std::cout << "It info after: S " << it - tmp_fix << " E " << tmp_fix + size - it << std::endl;
        }

        hipFree(predicate);
        hipFree(to_fix_cpy);
    }
} // namespace CustomCore
