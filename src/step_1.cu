#include "hip/hip_runtime.h"
#include "fix_gpu.hh"

#include <iostream>
#include <thrust/scan.h>
#include <thrust/execution_policy.h>
#include <thrust/copy.h>

namespace Core
{
    struct generate_mask
    {
        __host__ __device__ bool operator()(const int x)
        {
            return x != -27;
        }
    };

    // Compact
    void step_1(thrust::device_vector<int> &to_fix)
    {
        thrust::copy_if(to_fix.begin(), to_fix.end(), to_fix.begin(), generate_mask());
    }

} // namespace Core

namespace CustomCore
{
    __global__ void build_predicate2(int *to_fix, int *predicate, int size)
    {
        // WIP TODO
        int id = (blockIdx.x * blockDim.x + threadIdx.x);

        // Grid stride loop pattern and vectorial access
        for (int i = id; i < size / 4; i += blockDim.x * gridDim.x)
        {
            int4 vals = reinterpret_cast<int4 *>(to_fix)[i];
            predicate[i] = vals.x != -27;
            predicate[i + 1] = vals.y != -27;
            predicate[i + 2] = vals.z != -27;
            predicate[i + 3] = vals.w != -27;
        }

        int remainder = size % 4;
        if (id == size % 4 && remainder != 0)
        {
            while (remainder)
            {
                int idx = size - remainder--;
                predicate[idx] = to_fix[idx] != -27;
            }
        }
    }
    __global__ void build_predicate1(int *to_fix, int *predicate, int size)
    {
        int id = blockIdx.x * blockDim.x + threadIdx.x;
        // Grid stride loop pattern
        for (int i = id; i < size; i += blockDim.x * gridDim.x)
            predicate[i] = to_fix[i] != -27;
    }

    __global__ void build_predicate0(int *to_fix, int *predicate, int size)
    {
        int id = blockIdx.x * blockDim.x + threadIdx.x;
        if (id < size)
            predicate[id] = to_fix[id] != -27;
    }

    __global__ void scatter1(int *to_fix, int *predicate, int size)
    {
        int id = blockIdx.x * blockDim.x + threadIdx.x;
        if (id < size)
        {
            int val = to_fix[id];
            __syncthreads();
            if (val != -27)
                to_fix[predicate[id]] = val;
        }
    }

    __global__ void scatter0(int *to_fix, int *to_fix_cpy, int *predicate, int size)
    {
        int id = blockIdx.x * blockDim.x + threadIdx.x;
        if (id < size)
        {
            int val = to_fix_cpy[id];
            if (val != -27)
                to_fix[predicate[id]] = val;
        }
    }

    // Compact
    void step_1(int *to_fix, ImageInfo imageInfo)
    {
        int size = imageInfo.corrupted_size;
        int nbBlocks = std::ceil((float)size / NB_THREADS);

        // 1 Build the predicate vector
        int *predicate;
        cudaMalloc_custom(&predicate, sizeof(int) * size);
        build_predicate1<<<nbBlocks / 4, NB_THREADS>>>(to_fix, predicate, size);

        checkKernelError("build_predicate");

        // 2 Exclusive sum of the predicate
        std::cout << "Start scan" << std::endl;
        scan(predicate, size, false);

        // 3 Scatter to the corresponding addresses
        const int image_size = imageInfo.width * imageInfo.height;
        scatter1<<<nbBlocks, NB_THREADS>>>(to_fix, predicate, size);

        checkKernelError("scatter");

        hipFree(predicate);
    }
} // namespace CustomCore
