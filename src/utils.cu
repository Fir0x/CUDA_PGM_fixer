#include "fix_gpu.hh"

namespace CustomCore
{
    void cudaMalloc_custom(int **ptr, size_t size)
    {
        hipError_t err = hipMalloc(ptr, size);
        if (err != 0)
        {
            std::cout << "Malloc ERROR: " << hipGetErrorString(err) << std::endl;
            exit(err);
        }
    }

    void checkKernelError(std::string name)
    {
        hipError_t err = hipGetLastError();
        if (err != 0)
        {
            std::cout << "Kernel ERROR: " << name << ": " << hipGetErrorString(err) << std::endl;
            exit(err);
        }
    }
}