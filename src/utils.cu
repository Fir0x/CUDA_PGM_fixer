#include "fix_gpu.hh"

namespace CustomCore
{
    void cudaMalloc_custom(int **ptr, size_t size, int line,const char* file)
    {
        hipError_t err = hipMalloc(ptr, size);
        if (err != 0)
        {
            std::cout << "Malloc ERROR: " << hipGetErrorString(err) << " at " << file << ":" << line << std::endl;
            exit(err);
        }
    }

    void checkKernelError(std::string name)
    {
        hipError_t err = hipGetLastError();
        if (err != 0)
        {
            std::cout << "Kernel ERROR: " << name << ": " << hipGetErrorString(err) << std::endl;
            exit(err);
        }
    }
}