#include "fix_gpu.hh"

namespace CustomCore
{
    void cudaMalloc_custom(int **ptr, size_t size)
    {
        hipError_t err = hipMalloc(ptr, size);
        if (err != 0)
        {
            std::cout << "Malloc ERROR: " << hipGetErrorString(err) << std::endl;
            exit(err);
        }
    }

    void cudaMallocAsync_custom(int **ptr, size_t size, hipStream_t stream)
    {
        hipError_t err = hipMallocAsync(ptr, size, stream);
        if (err != 0)
        {
            std::cout << "Malloc ERROR: " << hipGetErrorString(err) << std::endl;
            exit(err);
        }
    }

    void checkKernelError(std::string name)
    {
        hipError_t err = hipGetLastError();
        if (err != 0)
        {
            std::cout << "Kernel ERROR: " << name << ": " << hipGetErrorString(err) << std::endl;
            exit(err);
        }
    }
}